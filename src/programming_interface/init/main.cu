#include <cassert>
#include <cstdio>

#include <hip/hip_runtime.h>

int main() {
    // Instruct CUDA to yield its thread when waiting for results from the device.
    unsigned int device_flags = hipDeviceScheduleYield;
    // Tell the CUDA runtime that DeviceFlags is being set in cudaInitDevice call 
    unsigned int flags = cudaInitDeviceFlagsAreValid;
    // Initialize device to be used for GPU executions.
    auto e = cudaInitDevice(0, device_flags, flags);
    assert(e == hipSuccess);

    // Set device to be used for GPU executions.
    e = hipSetDevice(0);
    assert(e == hipSuccess);
    
    int device;
    e = hipGetDevice(&device);
    assert(e == hipSuccess);
    assert(device == 0);

    unsigned int current_flags;
    e = hipGetDeviceFlags(&current_flags);
    assert(e == hipSuccess);
    // Flags returned by this function may specifically include cudaDeviceMapHost
    // even though it is not accepted by cudaSetDeviceFlags because it is implicit
    // in runtime API flags.
    assert((current_flags & device_flags) == device_flags);
}
