#include <hip/hip_runtime.h>

__constant__ float constData[256];
__device__ float   devData;
__device__ float*  devPointer;

int main() {
  float data[256];
  hipMemcpyToSymbol(HIP_SYMBOL(constData), data, sizeof(data));
  hipMemcpyFromSymbol(data, HIP_SYMBOL(constData), sizeof(data));

  float value = 3.14f;
  hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float));

  float* ptr;
  hipMalloc(&ptr, 256 * sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(devPointer), &ptr, sizeof(ptr));

  hipFree(ptr);
}
