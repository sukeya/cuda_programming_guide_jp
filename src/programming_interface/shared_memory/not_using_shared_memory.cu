#include <cassert>
#include <memory>

// CUDA runtime
#include <hip/hip_runtime.h>

// Thread block size
#define BLOCK_SIZE 16

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
struct Matrix {
  int    width;
  int    height;
  float* elements;
};

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C) {
  // Load A and B to device memory
  Matrix d_A;
  d_A.width   = A.width;
  d_A.height  = A.height;
  size_t size = A.width * A.height * sizeof(float);
  hipMalloc(&d_A.elements, size);
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
  Matrix d_B;
  d_B.width  = B.width;
  d_B.height = B.height;
  size       = B.width * B.height * sizeof(float);
  hipMalloc(&d_B.elements, size);
  hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

  // Allocate C in device memory
  Matrix d_C;
  d_C.width  = C.width;
  d_C.height = C.height;
  size       = C.width * C.height * sizeof(float);
  hipMalloc(&d_C.elements, size);

  // Invoke kernel
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
  MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

  // Read C from device memory
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
// Calculate C = A * B
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
  // Each thread computes one element of C
  // by accumulating results into Cvalue
  float Cvalue = 0;
  int   row    = blockIdx.y * blockDim.y + threadIdx.y;
  int   col    = blockIdx.x * blockDim.x + threadIdx.x;
  for (int e = 0; e < A.width; ++e)
    Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
  C.elements[row * C.width + col] = Cvalue;
}

int main(int argc, char** argv) {
  int cols = 3 * BLOCK_SIZE;
  int rows = 3 * BLOCK_SIZE;

  auto A = std::make_unique<float[]>(rows * cols);
  auto B = std::make_unique<float[]>(rows * cols);

  for (int i = 0; i < rows; ++i) {
    for (int j = 0; j < cols; ++j) {
      A[i * cols + j] = i * j;
    }
  }

  for (int i = 0; i < rows; ++i) {
    for (int j = 0; j < cols; ++j) {
      B[i * cols + j] = i + j;
    }
  }

  auto A_cuda = Matrix{.width = cols, .height = rows, .elements = A.get()};
  auto B_cuda = Matrix{.width = cols, .height = rows, .elements = B.get()};

  auto C      = std::make_unique<float[]>(rows * cols);
  auto C_cuda = Matrix{.width = cols, .height = rows, .elements = C.get()};

  MatMul(A_cuda, B_cuda, C_cuda);

  for (int i = 0; i < rows; ++i) {
    for (int j = 0; j < cols; ++j) {
      auto answer = 0.0f;
      for (int k = 0; k < rows; ++k) {
        answer += (i * k) * (k + j);
      }
      assert(C[i * cols + j] == answer);
    }
  }

  return 0;
}