#include "hip/hip_runtime.h"
#include <cstdio>
#include <memory>
#include <random>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

/*
 * 分散共有メモリを用いてヒストグラムを計算するカーネル
 *
 * 方針:
 * 全てのビンを各スレッドブロックで分けて共有メモリに保持し、計算が全部終わったらグローバルメモリに書き込む。
 * 入力はスレッド数でmodを取り、自身のスレッドIDの同値類のそれぞれの値を配列のインデックスと見てビンに足す。
*/
__global__ void clusterHist_kernel(
    int *bins,
    const int nbins,
    const int bins_per_block,
    const int *__restrict__ input,
    size_t array_size
) {
  // 共有メモリを動的に確保するため、extern宣言する
  extern __shared__ int smem[];

  namespace cg = cooperative_groups;

  int tid = cg::this_grid().thread_rank();

  // Cluster initialization, size and calculating local bin offsets.
  cg::cluster_group cluster = cg::this_cluster();
  int cluster_size = cluster.dim_blocks().x;

  for (int i = threadIdx.x; i < bins_per_block; i += blockDim.x)
  {
    smem[i] = 0; //Initialize shared memory histogram to zeros
  }

  // cluster synchronization ensures that shared memory is initialized to zero in
  // all thread blocks in the cluster. It also ensures that all thread blocks
  // have started executing and they exist concurrently.
  cluster.sync();

  for (int i = tid; i < array_size; i += blockDim.x * gridDim.x)
  {
    int ldata = input[i];

    //Find the right histogram bin.
    int binid = ldata;
    if (ldata < 0) {
      binid = 0;
    } else if (ldata >= nbins) {
      binid = nbins - 1;
    }

    //Find destination block rank and offset for computing
    //distributed shared memory histogram
    int dst_block_rank = (int)(binid / bins_per_block);
    int dst_offset = binid % bins_per_block;

    //Pointer to target block shared memory
    int *dst_smem = cluster.map_shared_rank(smem, dst_block_rank);

    //Perform atomic update of the histogram bin
    atomicAdd(dst_smem + dst_offset, 1);
  }

  // cluster synchronization is required to ensure all distributed shared
  // memory operations are completed and no thread block exits while
  // other thread blocks are still accessing distributed shared memory
  cluster.sync();

  // Perform global memory histogram, using the local distributed memory histogram
  for (int i = threadIdx.x; i < bins_per_block; i += blockDim.x)
  {
    atomicAdd(&bins[i], smem[i]);
  }
}


int main() {
  constexpr int array_size = 64;
  constexpr int threads_per_block = 16;
  constexpr int nbins = 16;

  hipLaunchConfig_t config = {0};
  config.gridDim = array_size / threads_per_block;
  config.blockDim = threads_per_block;

  // cluster_size depends on the histogram size.
  // ( cluster_size == 1 ) implies no distributed shared memory,
  // just thread block local shared memory
  int cluster_size = 2; // size 2 is an example here
  int nbins_per_block = nbins / cluster_size;

  //dynamic shared memory size is per block.
  //Distributed shared memory size =  cluster_size * nbins_per_block * sizeof(int)
  config.dynamicSmemBytes = nbins_per_block * sizeof(int);

  // 動的共有メモリの最大サイズを変更
  auto e = hipFuncSetAttribute(
    (void *)clusterHist_kernel,
    hipFuncAttributeMaxDynamicSharedMemorySize,
    config.dynamicSmemBytes
  );
  assert(e == hipSuccess);

  hipLaunchAttribute attribute[1];
  attribute[0].id = cudaLaunchAttributeClusterDimension;
  attribute[0].val.clusterDim.x = cluster_size;
  attribute[0].val.clusterDim.y = 1;
  attribute[0].val.clusterDim.z = 1;

  config.numAttrs = 1;
  config.attrs = attribute;

  int* dev_bins;

  e = hipMalloc(&dev_bins, nbins * sizeof(int));
  assert(e == hipSuccess);

  std::vector<int> input;
  std::random_device seed_gen;
  std::mt19937 engine(seed_gen());

  // 範囲外のチェックをしているか確認するために、わざと範囲外を生成するように指定
  std::uniform_int_distribution<int> dist(-1, nbins);

  for (std::size_t i = 0; i < array_size; ++i) {
    input.push_back(dist(engine));
  }

  int* dev_input;
  e = hipMalloc(&dev_input, input.size() * sizeof(int));
  assert(e == hipSuccess);

  e = hipMemcpy(dev_input, input.data(), input.size() * sizeof(int), hipMemcpyHostToDevice);
  assert(e == hipSuccess);

  e = cudaLaunchKernelEx(
    &config,
    clusterHist_kernel,
    dev_bins,
    nbins,
    nbins_per_block,
    dev_input,
    array_size
  );
  assert(e == hipSuccess);

  std::unique_ptr<int[]> bins = std::make_unique<int[]>(nbins);

  e = hipMemcpy(bins.get(), dev_bins, nbins * sizeof(int), hipMemcpyDeviceToHost);
  assert(e == hipSuccess);

  std::vector<int> answer;
  answer.resize(nbins, 0);
  for (const auto i : input) {
    if (i < 0) {
      ++answer.front();
    } else if (i >= array_size) {
      ++answer.back();
    } else {
      ++answer[i];
    }
  }

  for (int i = 0; i < nbins; ++i) {
    assert(answer[i] == bins[i]);
  }

  e = hipFree(dev_input);
  assert(e == hipSuccess);
  e = hipFree(dev_bins);
  assert(e == hipSuccess);
}
